#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <stdexcept>
#include <cstdio>
#include <torch/extension.h>

// cublas API error checking
#define CUBLAS_CHECK(err)                                                                          \
    do {                                                                                           \
        hipblasStatus_t err_ = (err);                                                               \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                                       \
            std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__);                   \
            throw std::runtime_error("cublas error");                                              \
        }                                                                                          \
    } while (0)

bool cublas_kernel(__half* A, __half* B, __half* D, int M, int N, int K) {
	float alpha = 1;
	float beta = 0;

	hipblasHandle_t handle;
	CUBLAS_CHECK(hipblasCreate(&handle));
	
	CUBLAS_CHECK(
		hipblasGemmEx(handle,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			N, M, K,
			&alpha,
			B, HIP_R_16F, K,
			A, HIP_R_16F, K,
			&beta,
			D, HIP_R_16F, N,
			HIP_R_32F,
			HIPBLAS_GEMM_DEFAULT
		)
	);
	
	CUBLAS_CHECK(hipblasDestroy(handle));
	return true;
}

bool cublas_tn(const torch::Tensor& X, const torch::Tensor& W, const torch::Tensor& Y) {
	half* X_ = reinterpret_cast<half*>(X.data_ptr());
	half* W_ = reinterpret_cast<half*>(W.data_ptr());
	half* Y_ = reinterpret_cast<half*>(Y.data_ptr());

	return cublas_kernel(X_, W_, Y_, X.size(0), W.size(0), X.size(1));
}
