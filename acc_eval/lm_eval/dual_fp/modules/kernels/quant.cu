#include "hip/hip_runtime.h"
#include <assert.h>
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>

#include "e4m3.h"
#include "e5m2.h"
#include "e4m3_scale.h"

void gemm_e4m3(torch::Tensor X, torch::Tensor W, torch::Tensor Y) {
	cutlass::float_e4m3_t* X_ = reinterpret_cast<cutlass::float_e4m3_t*>(X.data_ptr());
	cutlass::float_e4m3_t* W_ = reinterpret_cast<cutlass::float_e4m3_t*>(W.data_ptr());
	half* Y_ = reinterpret_cast<half*>(Y.data_ptr());

	e4m3(X_, W_, Y_, X.size(0), W.size(1), X.size(1));
}

void gemm_e5m2(torch::Tensor X, torch::Tensor W, torch::Tensor Y) {
        cutlass::float_e5m2_t* X_ = reinterpret_cast<cutlass::float_e5m2_t*>(X.data_ptr());
        cutlass::float_e5m2_t* W_ = reinterpret_cast<cutlass::float_e5m2_t*>(W.data_ptr());
        half* Y_ = reinterpret_cast<half*>(Y.data_ptr());

        e5m2(X_, W_, Y_, X.size(0), W.size(1), X.size(1));
}

void gemm_e4m3_scale(torch::Tensor X, torch::Tensor W, torch::Tensor Y) {
        cutlass::float_e4m3_t* X_ = reinterpret_cast<cutlass::float_e4m3_t*>(X.data_ptr());
        cutlass::float_e4m3_t* W_ = reinterpret_cast<cutlass::float_e4m3_t*>(W.data_ptr());
        half* Y_ = reinterpret_cast<half*>(Y.data_ptr());

        e4m3_scale(X_, W_, Y_, X.size(0), W.size(1), X.size(1));
}

__global__ void E4M3_ROUND_TO_NEAREST(half* SRC, cutlass::float_e4m3_t* DEST, int S) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        if (tid >= S) return;

        DEST[tid] = cutlass::float_e4m3_t::from_half(SRC[tid]);
}

void quant_e4m3(torch::Tensor A, torch::Tensor A_) {
        int D1 = A.sizes()[0];
        int D2 = A.sizes()[1];

        half* SRC_A = reinterpret_cast<half*>(A.data_ptr());
        cutlass::float_e4m3_t* DEST_A = reinterpret_cast<cutlass::float_e4m3_t*>(A_.data_ptr());

        int BLK_SZ = 256;
        dim3 blockDim;
        dim3 gridDim;

        blockDim.x = BLK_SZ;
        gridDim.x = (D1 * D2 + BLK_SZ - 1) / BLK_SZ;
        E4M3_ROUND_TO_NEAREST<<<gridDim, blockDim>>>(SRC_A, DEST_A, D1 * D2);
}

__global__ void E5M2_ROUND_TO_NEAREST(half* SRC, cutlass::float_e5m2_t* DEST, int S) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        if (tid >= S) return;

        DEST[tid] = cutlass::float_e5m2_t::from_half(SRC[tid]);
}

void quant_e5m2(torch::Tensor A, torch::Tensor A_) {
        int D1 = A.sizes()[0];
        int D2 = A.sizes()[1];

        half* SRC_A = reinterpret_cast<half*>(A.data_ptr());
        cutlass::float_e5m2_t* DEST_A = reinterpret_cast<cutlass::float_e5m2_t*>(A_.data_ptr());

        int BLK_SZ = 256;
        dim3 blockDim;
        dim3 gridDim;

        blockDim.x = BLK_SZ;
        gridDim.x = (D1 * D2 + BLK_SZ - 1) / BLK_SZ;
        E5M2_ROUND_TO_NEAREST<<<gridDim, blockDim>>>(SRC_A, DEST_A, D1 * D2);
}

__global__ void FP8_ROUND_TOWARD_ZERO(half* SRC, cutlass::float_e4m3_t* DEST, int S) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= S) return;

	half h = SRC[tid];
	uint16_t a = reinterpret_cast<uint16_t const &>(h);
	uint8_t s1 = (a >> 15) & 0x1;
	uint8_t e1 = (a >> 10) & 0xf;
	uint8_t m1 = (a >> 7) & 0x7;
	uint8_t m2 = a & 0x7f;
	uint8_t b = (s1 << 7) | (e1 << 3) | (m1 << 0);

	assert(((a >> 14) & 0x1) == 0);

	if (e1 == 15 && m1 == 6) {
		DEST[tid] = cutlass::float_e4m3_t::bitcast(b);
		return;
	}

	if ((m2 > 64) || ((m2 == 64) && ((m1 & 0x1) == 1))) DEST[tid] = cutlass::float_e4m3_t::bitcast(b + 1);
	else DEST[tid] = cutlass::float_e4m3_t::bitcast(b);
}

void quant_e4m3_scale(torch::Tensor A, torch::Tensor A_) {
        int D1 = A.sizes()[0];
	int D2 = A.sizes()[1];

        half* SRC_A = reinterpret_cast<half*>(A.data_ptr());
        cutlass::float_e4m3_t* DEST_A = reinterpret_cast<cutlass::float_e4m3_t*>(A_.data_ptr());
        
        int BLK_SZ = 256;
        dim3 blockDim;
        dim3 gridDim;

        blockDim.x = BLK_SZ;
        gridDim.x = (D1 * D2 + BLK_SZ - 1) / BLK_SZ;
        FP8_ROUND_TOWARD_ZERO<<<gridDim, blockDim>>>(SRC_A, DEST_A, D1 * D2);
}

PYBIND11_MODULE(dual_fp_ext, m) {
    m.def("gemm_e4m3", &gemm_e4m3);
    m.def("gemm_e5m2", &gemm_e5m2);
    m.def("gemm_e4m3_scale", &gemm_e4m3_scale);
    m.def("quant_e4m3", &quant_e4m3);
    m.def("quant_e5m2", &quant_e5m2);
    m.def("quant_e4m3_scale", &quant_e4m3_scale);
}
