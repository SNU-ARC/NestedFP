#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include "cutlass/cutlass.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/util/packed_stride.hpp"

using MainloopScheduleType = cutlass::gemm::KernelTmaWarpSpecialized;
using EpilogueScheduleType = cutlass::epilogue::TmaWarpSpecialized;
using TileSchedulerType = cutlass::gemm::PersistentScheduler;

using namespace cute;

/* =========================================================
 * 1.  CUDA‑graph–safe context (out‑variant)
 *      ‑ caller supplies tensor D
 * ===================================================== */
template<int T1, int T2, int T3>
class TmaCtx_out {
 public:
  /* ---------- type aliases ----------------------------------- */
  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
          cute::Shape<cute::Int<T1>, cute::Int<T2>, cute::Int<T3>>,
          cute::Shape<cute::_1, cute::_1, cute::_1>,
          cutlass::epilogue::collective::EpilogueTileAuto,
          float, float,
          cutlass::half_t, cutlass::layout::ColumnMajor, 8,
          cutlass::half_t, cutlass::layout::ColumnMajor, 8,
          EpilogueScheduleType>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
          cutlass::half_t, cutlass::layout::RowMajor,    8,
          cutlass::half_t, cutlass::layout::ColumnMajor, 8,
          float,
          cute::Shape<cute::Int<T1>, cute::Int<T2>, cute::Int<T3>>,
          cute::Shape<cute::_1,  cute::_1,  cute::_1>,
          cutlass::gemm::collective::StageCountAutoCarveout<
              static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
          MainloopScheduleType>::CollectiveOp;

  using GemmKernel     = cutlass::gemm::kernel::GemmUniversal<
                           cute::Shape<int,int,int,int>,
                           CollectiveMainloop,
                           CollectiveEpilogue,
                           TileSchedulerType>;
  using DeviceKernel   = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  using ElementCompute = typename DeviceKernel::EpilogueOutputOp::ElementCompute;

  using StrideA = typename DeviceKernel::GemmKernel::StrideA;
  using StrideB = typename DeviceKernel::GemmKernel::StrideB;
  using StrideC = typename DeviceKernel::GemmKernel::StrideC;
  using StrideD = typename DeviceKernel::GemmKernel::StrideD;

  /* ---------- public API ------------------------------------- */
  void initialize(const at::Tensor& A,
                  const at::Tensor& B,
                  const at::Tensor& D) {
    queryShapes(A,B,D);
    createWorkspace(A.device());     // workspace only
    buildArguments(A,B,D);
    initialized = true;
  }

  /* 재초기화 ― shape(M,N,K) 변동 시 */
  void maybe_reinitialize(const at::Tensor& A,
                          const at::Tensor& B,
                          const at::Tensor& D) {
    if (!initialized || !sameShape(A,B,D)) {
      initialize(A,B,D);
    }
  }

  /* 단순 ptr 갱신 */
  void refresh_arguments(const at::Tensor& A,
                         const at::Tensor& B,
                         const at::Tensor& D) {
    buildArguments(A,B,D);
  }

  void run(hipStream_t stream,
           const at::Tensor& A,
           const at::Tensor& B,
           const at::Tensor& D) {
    refresh_arguments(A,B,D);
    auto status = gemm_op.run(args,
                              workspace.data_ptr<uint8_t>(),
                              stream);
    TORCH_CHECK(status == cutlass::Status::kSuccess,
                "CUTLASS GEMM failed");
  }

  bool sameShape(const at::Tensor& A,
                 const at::Tensor& B,
                 const at::Tensor& D) const {
    return (A.size(0)==M && B.size(0)==N &&
            A.size(1)==K &&        // K
            D.size(0)==M && D.size(1)==N);
  }

 private:
  /* ---------- helpers ---------------------------------------- */
  void queryShapes(const at::Tensor& A,
                   const at::Tensor& B,
                   const at::Tensor& /*D*/) {
    M = static_cast<int>(A.size(0));
    N = static_cast<int>(B.size(0));
    K = static_cast<int>(A.size(1));
  }

  void createWorkspace(const at::Device& dev) {
    size_t ws_bytes =
        DeviceKernel::get_workspace_size(
            typename DeviceKernel::Arguments{});
    workspace = at::empty({static_cast<long>(ws_bytes)},
                          at::dtype(at::kByte).device(dev));
  }

  void buildArguments(const at::Tensor& A,
                      const at::Tensor& B,
                      const at::Tensor& D) {
    const cutlass::half_t* A_ =
        reinterpret_cast<const cutlass::half_t*>(A.contiguous().data_ptr());
    const cutlass::half_t* B_ =
        reinterpret_cast<const cutlass::half_t*>(B.contiguous().data_ptr());
    cutlass::half_t* D_ =
        reinterpret_cast<cutlass::half_t*>(D.data_ptr());

    int dev_id; hipGetDevice(&dev_id);
    hw_info.device_id = dev_id;
    hw_info.sm_count =
        cutlass::KernelHardwareInfo::query_device_multiprocessor_count(dev_id);

    args = typename DeviceKernel::Arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K, /*L=*/1},

      /* A, B (packed row/col) */
      {
        A_,
        cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, 1)),
        B_,
        cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, 1))
      },

      /* epilogue (C=null, α=1, β=0) */
      {
        {ElementCompute(1.f), ElementCompute(0.f)},
        nullptr,
        cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, 1)),
        D_,
        cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, 1))
      },

      hw_info
    };
  }

  /* ---------- data members ----------------------------------- */
  bool                         initialized{false};
  int                          M{0}, N{0}, K{0};
  at::Tensor                   workspace;
  typename DeviceKernel::Arguments args;
  DeviceKernel                 gemm_op;
  cutlass::KernelHardwareInfo  hw_info;
};

/* =========================================================
 * 2.  PyBind 노출 함수 ( *_out 스타일)
 *      ‑ returns void, writes into caller‑supplied D
 * ===================================================== */
template<int T1,int T2,int T3>
void cutlass_tma_warp_specialized_kernel(const at::Tensor& A,
                                             const at::Tensor& B,
                                            at::Tensor& D) {
  TORCH_CHECK(A.device().is_cuda() && B.device().is_cuda() && D.device().is_cuda(),
              "All tensors must be CUDA tensors");
  TORCH_CHECK(A.dtype() == torch::kF16 && B.dtype() == torch::kF16 && D.dtype() == torch::kF16,
              "All tensors must be torch.float16");
  TORCH_CHECK(A.size(1) == B.size(1), "K dimension mismatch (A.cols vs B.cols)");
  TORCH_CHECK(D.is_contiguous(),      "Output D must be contiguous");
  TORCH_CHECK(D.size(0) == A.size(0) && D.size(1) == B.size(0),
              "D must have shape (M, N)");

  /* thread‑local ctx: host‑thread당 하나 → graph‑safe */
  static thread_local TmaCtx_out<T1,T2,T3> ctx;

  ctx.maybe_reinitialize(A,B,D);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ctx.run(stream, A, B, D);
}

template void cutlass_tma_warp_specialized_kernel<64, 16, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 16, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 16, 256>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 32, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 32, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 32, 256>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 64, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 64, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 64, 256>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 128, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 128, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 128, 256>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 256, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<64, 256, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 16, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 16, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 16, 256>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 32, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 32, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 32, 256>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 64, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 64, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 64, 256>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 128, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 128, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 256, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<128, 256, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<256, 16, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<256, 16, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<256, 32, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<256, 32, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<256, 64, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<256, 64, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<256, 128, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<256, 128, 128>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
template void cutlass_tma_warp_specialized_kernel<256, 256, 64>(const at::Tensor& A, const at::Tensor& B, at::Tensor& D);
